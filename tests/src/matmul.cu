#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define BLOCK_SIZE 16

typedef struct {
    int width;
    int height;
    float * elements;
} Matrix;

__global__ void matMulNaiveKernel(Matrix A, Matrix B, Matrix C){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float Cvalue;
    for(int i = 0; i< A.width; i++){
        Cvalue += A.elements[row * A.width + i] * B.elements[i * B.width + col];
    }
    C.elements[row * C.width + col] = Cvalue;
}



void matMul(const Matrix A, const Matrix B, Matrix C){

    // Allocate device mem
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    checkCudaErrors(hipMalloc((void **)&d_A.elements, sizeof(float)*A.width*A.height));

    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    checkCudaErrors(hipMalloc((void **)&d_B.elements, sizeof(float)*B.width*B.height));

    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    checkCudaErrors(hipMalloc((void **)&d_C.elements, sizeof(float)*C.width*C.height));

    // Transfer mem to device
    checkCudaErrors(hipMemcpy(d_A.elements, A.elements, sizeof(float)*A.width*A.height, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B.elements, B.elements, sizeof(float)*B.width*B.height, hipMemcpyHostToDevice));

    // Initialise CUDA timing
    float milliGPU;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width/dimBlock.x, A.height/dimBlock.y);

    hipEventRecord(start);

    matMulNaiveKernel<<<dimGrid, dimBlock >>>(d_A, d_B, d_C);
    getLastCudaError("Exec of matMulKernel failed \n");

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliGPU, start, stop);
    printf("GPU execution time (ms): %f \n", milliGPU);

    // Copy mem device to host
    checkCudaErrors(hipMemcpy(C.elements, d_C.elements, sizeof(float)*C.width*C.height, hipMemcpyDeviceToHost));

    // Free device mem
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

}

int main(){

    // Allocate A in host
    Matrix A;
    A.width = 1000;
    A.height = 2000;
    A.elements = (float *)malloc(sizeof(float)*A.width*A.height);

    for(int i=0; i<A.height*A.width; i++){
        A.elements[i] = i;
    }

    // Allocate B in host
    Matrix B;
    B.width = 2000;
    B.height = 500;
    B.elements = (float *)malloc(sizeof(float)*B.width*B.height);

    for(int i=0; i<B.height*B.width; i++){
        B.elements[i] = i;
    }

    // Allocate C in host
    Matrix C;
    C.height = A.height;
    C.width = B.width;
    C.elements = (float *)malloc(sizeof(float)*C.width*C.height);

    matMul(A, B, C);

    // Free mem in host
    free(A.elements);
    free(B.elements);

    return 0;
}












